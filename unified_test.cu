#include <iostream>
#include <hip/hip_runtime.h> 
using namespace std;

// Derived class
class Rectangle
{
   public:
      Rectangle()
      {
	hipMallocManaged(&width, sizeof(int));
	hipMallocManaged(&height, sizeof(int));
        //width = (int *)malloc(sizeof(int));
        //height = (int *)malloc(sizeof(int));
      }
      int getArea()
      { 
         return (*width * *height); 
      }
      int* width;
      int* height;
};

// Base class
class Shape 
{
   public:
    Shape()
    {
 	hipMallocManaged(&rect, sizeof(Rectangle));	
        *(rect->width) = 20;
        *(rect->height) = 10;
    }
    Rectangle* rect;
};

__global__ void change_width(Shape* sha)
{
	*(sha->rect->width) = 10;
	return;
}

int main(void)
{
   Shape* sha;
   hipMallocManaged(&sha,sizeof(Shape));
   //Shape* sha = new Shape(); 
   change_width<<<1,1,0>>>(sha);
   hipDeviceSynchronize(); 
   // Print the area of the object.
   cout << "Total area: " << sha->rect->getArea() << endl;

   return 0;
}
