
#include <hip/hip_runtime.h>
#include <iostream>
 
using namespace std;

// Derived class
class Rectangle
{
   public:
      Rectangle()
      {
        width = (int *)malloc(sizeof(int));
        height = (int *)malloc(sizeof(int));
      }
      int getArea()
      { 
         return (*width * *height); 
      }
      int* width;
      int* height;
};
// Base class
class Shape 
{
   public:
    Shape()
    {
        rect = new Rectangle();
        *(rect->width) = 20;
        *(rect->height) = 10;
    }
    Rectangle* rect;
};


int main(void)
{
    Shape* sha = new Shape(); 

   // Print the area of the object.
   cout << "Total area: " << sha->rect->getArea() << endl;

   return 0;
}
