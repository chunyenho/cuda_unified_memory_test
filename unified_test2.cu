#include <iostream>
#include <hip/hip_runtime.h> 
using namespace std;

// Derived class
class Rectangle
{
    public:
        //don't use constructor
        Rectangle()
        {
            // width = (int *)malloc(sizeof(int));
            // height = (int *)malloc(sizeof(int));  
            // *width = 10;
            // *height = 30;
        }
        void build_function()
        {
            hipMallocManaged(&width,sizeof(int));
            hipMallocManaged(&height,sizeof(int));
            *width = 10;
            *height = 30;
        }
        __device__ __host__ void getArea()
        { 
            product = *width * *height; 
        }
        int product;
    private:
        int* width;
        int* height;
};

// Base class
class Shape 
{
    public:
        //don't use constructor
        Shape()
        {
            // rect = new Rectangle();
        }

        void build_function()
        {
            hipMallocManaged(&rect,sizeof(Rectangle));
            rect->build_function();   
        }
        Rectangle* rect;
};

__global__ void kernel(Shape* sha)
{
    sha->rect->getArea(); 
}

int main(void)
{
    //    Shape* sha = new Shape();
    Shape* sha;
    hipMallocManaged(&sha,sizeof(Shape));
    sha->build_function();

    kernel<<<1,1,0>>>(sha);
    hipDeviceSynchronize(); 
    cout << "Total area: " << sha->rect->product << endl;
    return 0;
}
